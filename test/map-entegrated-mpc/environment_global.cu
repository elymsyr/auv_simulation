#include "hip/hip_runtime.h"
#include "environment.h"
#include <chrono>
#include <algorithm>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cfloat>

// Map
__global__ void slidePhase1(uint8_t* grid, uint8_t* tempGrid, int width, int height, int2 shift) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (tx >= width || ty >= height) return;

    int dst_idx = ty * width + tx;
    int src_x = tx + shift.x;
    int src_y = ty + shift.y;

    if (src_x >= 0 && src_x < width && src_y >= 0 && src_y < height) {
        tempGrid[dst_idx] = grid[src_y * width + src_x];
    } else {
        tempGrid[dst_idx] = 0;
    }
}

__global__ void slidePhase2(uint8_t* grid, uint8_t* tempGrid, int width, int height) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (tx >= width || ty >= height) return;

    int idx = ty * width + tx;
    grid[idx] = tempGrid[idx];
}

__global__ void pointUpdateKernel(uint8_t* grid, int width, int height, float x_r, float y_r, float r_m, float2* coords_dev, uint8_t* values_dev, int count) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= count) return;

    float2 coord = coords_dev[tid];
    uint8_t val = values_dev[tid];

    int x_coor = static_cast<int>((coord.x - x_r) / r_m + width / 2.0f);
    int y_coor = static_cast<int>((coord.y - y_r) / r_m + height / 2.0f);

    if (x_coor >= 0 && x_coor < width && y_coor >= 0 && y_coor < height) {
        grid[y_coor * width + x_coor] = val;
    }
}

__global__ void singlePointUpdateKernel(uint8_t* grid, int width, int height, 
                                        float x_r, float y_r, float r_m,
                                        float world_x, float world_y, 
                                        uint8_t value, int radius, char mode) {
    if (mode == 'w') {
        world_x = (world_x - x_r) / r_m + width / 2.0f;
        world_y = (world_y - y_r) / r_m + height / 2.0f;
    }

    int x_coor = __float2int_rd(world_x);
    int y_coor = __float2int_rd(world_y);

    if (x_coor >= 0 && x_coor < width && y_coor >= 0 && y_coor < height) {
        int index = y_coor * width + x_coor;
        grid[index] = value;
    }

    if (value >= 250) {
        for (int di = -radius; di <= radius; di++) {
            for (int dj = -radius; dj <= radius; dj++) {
                // Calculate squared distance to avoid sqrt
                if (di * di + dj * dj <= radius * radius) {
                    int ni = y_coor + di;
                    int nj = x_coor + dj;
                    if (ni >= 0 && ni < height && nj >= 0 && nj < width) {
                        int n_index = ni * width + nj;
                        if (grid[n_index] < 250) {
                            grid[n_index] = value;
                        }
                    }
                }
            }
        }
    }
}

__global__ void obstacleSelectionKernel(uint8_t* grid, int width, int height, float wx, float wy, float* output_dists, float2* output_coords, int* output_count, int max_output, float circle_radius, float r_m_) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx >= width || idy >= height) return;
    
    int grid_idx = idy * width + idx;
    int cx = width / 2;
    int cy = height / 2;
    float dx = (float)(idx - cx);
    float dy = (float)(idy - cy);
    float dist = sqrtf(dx * dx + dy * dy) * r_m_;
    if (grid[grid_idx] >= 250 && (dist < circle_radius)) {
        int pos = atomicAdd(output_count, 1);
        if (pos < max_output) {
            output_dists[pos] = dist;
            output_coords[pos] = make_float2(wx + dx * r_m_, wy + dy * r_m_);
        }
        return;
    }
}

// A*
__device__ float atomicMinFloat(float* address, float val) {
    int* address_as_i = (int*)address;
    int old = *address_as_i;
    int expected;
    float old_val;
    do {
        expected = old;
        old_val = __int_as_float(expected);
        if (old_val <= val) return old_val;  // Return early if no update needed
        old = atomicCAS(address_as_i, expected, __float_as_int(val));
    } while (expected != old);
    return old_val;
}

__global__ void initKernel(Node* grid, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= width || idy >= height) return;

    int index = idy * width + idx;
    Node* node = &grid[index];
    node->x = idx;
    node->y = idy;
    node->g = FLT_MAX;
    node->h = FLT_MAX;
    node->parent_x = -1;
    node->parent_y = -1;
    node->status = 1;
}

__global__ void resetGridKernel(Node* grid, uint8_t* map, int width, int height, int goal_x, int goal_y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= width || idy >= height) return;

    int index = idy * width + idx;
    Node* node = &grid[index];
    
    node->parent_x = -1;
    node->parent_y = -1;
    
    if (idx == goal_x && idy == goal_y) {
        node->g = 0.0f;  // Goal cost is 0
        node->status = 1; // Goal is traversable
    } else {
        node->g = FLT_MAX;
        node->status = (map[index] >= 250) ? 0 : 1;
    }
    node->h = sqrtf(powf(idx - goal_x, 2) + powf(idy - goal_y, 2));
}

__global__ void wavefrontKernel(Node* grid, int width, int height, int* d_updated) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= width || idy >= height) return;
    
    int index = idy * width + idx;
    Node* node = &grid[index];
    
    // Only process free nodes that aren't the goal
    if (node->status != 1 || node->g == 0.0f) return;

    float min_g = FLT_MAX;
    int best_px = -1;
    int best_py = -1;

    // Check 8 neighbors
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            if (dx == 0 && dy == 0) continue;
            
            int nx = idx + dx;
            int ny = idy + dy;
            if (nx < 0 || nx >= width || ny < 0 || ny >= height) continue;
            
            int nidx = ny * width + nx;
            Node* neighbor = &grid[nidx];
            if (neighbor->status != 1) continue;  // Skip obstacles
            
            // Skip unvisited neighbors
            if (neighbor->g == FLT_MAX) continue;

            float cost = (dx != 0 && dy != 0) ? 1.4142f : 1.0f;
            float new_g = neighbor->g + cost;
            
            if (new_g < min_g) {
                min_g = new_g;
                best_px = nx;
                best_py = ny;
            }
        }
    }

    // Update if we found a better path
    if (min_g < node->g) {
        node->g = min_g;
        node->parent_x = best_px;
        node->parent_y = best_py;
        atomicOr(d_updated, 1);  // Mark update
    }
}

__global__ void reconstructPathKernel(Node* grid, int2* path, int* path_length, 
                                     int start_x, int start_y, int goal_x, int goal_y, 
                                     int width) {
    int x = start_x;
    int y = start_y;
    int count = 0;
    int max_length = width * width;  // Safeguard

    // Follow parent pointers from start to goal
    while (x != goal_x || y != goal_y) {
        if (count >= max_length) break;
        path[count++] = make_int2(x, y);
        
        int idx = y * width + x;
        int px = grid[idx].parent_x;
        int py = grid[idx].parent_y;
        
        if (px == -1 || py == -1) break;
        x = px;
        y = py;
    }
    
    // Add final goal point if reached
    if (x == goal_x && y == goal_y) {
        path[count++] = make_int2(x, y);
    }
    *path_length = count;
}